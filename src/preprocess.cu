#include "hip/hip_runtime.h"
// preprocess for detection
#include <opencv2/opencv.hpp>
#include <vector>

#include "common.hpp"
#include "yolo.hpp"
#include "stream.hpp"
#include "config.hpp"


using namespace google;

// warp affine transformation by bilinear interpolation
__global__ void warp_affine(
    uint8_t *src, int src_width, int src_height,
    uint8_t *dst, int dst_width, int dst_height,
    uint8_t fill_value, AffineMatrix M, int n)
{
    // int n = blockDim.z * blockIdx.z + threadIdx.z; // ibatch
    int offset_dst = n * dst_width * dst_height * 3;
    // int offset_src = n * src_width * src_height * 3;

    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;

    if (dx >= dst_width || dy >= dst_height)
        return;

    float c0 = fill_value, c1 = fill_value, c2 = fill_value;

    // multiply affine transformation matrix 
    float src_x = M.inv_mat[0] * dx + M.inv_mat[1] * dy + M.inv_mat[2];
    float src_y = M.inv_mat[3] * dx + M.inv_mat[4] * dy + M.inv_mat[5];

    // bilinear interpolation
    // if in range, do bilinear interpolation to get the RGB value
    // if out of range, fill with default RGB fill_value
    if (src_x >= -1 && src_x < src_width && src_y >= -1 && src_y < src_height)
    {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = ly * lx, w2 = ly * hx, w3 = hy * lx, w4 = hy * hx;
        uint8_t *v1 = const_values;
        uint8_t *v2 = const_values;
        uint8_t *v3 = const_values;
        uint8_t *v4 = const_values;
        if (y_low >= 0)
        {
            if (x_low >= 0)
                v1 = src + y_low * src_width * 3 + x_low * 3; // (x_low, y_low)

            if (x_high < src_width)
                v2 = src + y_low * src_width * 3 + x_high * 3; // (x_high, y_low)
        }

        if (y_high < src_height)
        {
            if (x_low >= 0)
                v3 = src + y_high * src_width * 3 + x_low * 3; // (x_low, y_high)

            if (x_high < src_width)
                v4 = src + y_high * src_width * 3 + x_high * 3; // (x_high, y_high)
        }

        c0 = roundf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0]);
        c1 = roundf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1]);
        c2 = roundf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2]);
    }

    uint8_t *pdst = dst + dy * dst_width * 3 + dx * 3 + offset_dst;
    pdst[0] = c0;
    pdst[1] = c1;
    pdst[2] = c2;
}

// batch warp affine transformation by bilinear interpolation
// this kernel function is DEPRECATED
// __global__ void batch_warp_affine(
//     uint8_t *src, int src_line_size, int src_width, int src_height,
//     uint8_t *dst, int dst_line_size, int dst_width, int dst_height,
//     uint8_t fill_value, AffineMatrix M)
// {
//     int n = blockDim.z * blockIdx.z + threadIdx.z; // ibatch
//     int offset_dst = n * dst_width * dst_height * 3;
//     int offset_src = n * src_width * src_height * 3;

//     int dx = blockDim.x * blockIdx.x + threadIdx.x;
//     int dy = blockDim.y * blockIdx.y + threadIdx.y;

//     if (dx >= dst_width || dy >= dst_height)
//         return;

//     float c0 = fill_value, c1 = fill_value, c2 = fill_value;
//     float src_x = 0;
//     float src_y = 0;
//     affine_project(M.inv_mat, dx, dy, &src_x, &src_y);

//     if (src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height)
//     {
//         // out of range
//         // when src_x < -1，high_x < 0，out of range
//         // when src_x >= -1，high_x >= 0，in range
//     }
//     else
//     {
//         int y_low = floorf(src_y);
//         int x_low = floorf(src_x);
//         int y_high = y_low + 1;
//         int x_high = x_low + 1;

//         uint8_t const_values[] = {fill_value, fill_value, fill_value};
//         float ly = src_y - y_low;
//         float lx = src_x - x_low;
//         float hy = 1 - ly;
//         float hx = 1 - lx;
//         float w1 = ly * lx, w2 = ly * hx, w3 = hy * lx, w4 = hy * hx;
//         uint8_t *v1 = const_values;
//         uint8_t *v2 = const_values;
//         uint8_t *v3 = const_values;
//         uint8_t *v4 = const_values;
//         if (y_low >= 0)
//         {
//             if (x_low >= 0)
//                 v1 = src + y_low * src_line_size + x_low * 3 + offset_src;

//             if (x_high < src_width)
//                 v2 = src + y_low * src_line_size + x_high * 3 + offset_src;
//         }

//         if (y_high < src_height)
//         {
//             if (x_low >= 0)
//                 v3 = src + y_high * src_line_size + x_low * 3 + offset_src;

//             if (x_high < src_width)
//                 v4 = src + y_high * src_line_size + x_high * 3 + offset_src;
//         }

//         c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
//         c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
//         c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
//     }

//     uint8_t *pdst = dst + dy * dst_line_size + dx * 3 + offset_dst;
//     pdst[0] = c0;
//     pdst[1] = c1;
//     pdst[2] = c2;
// }

// transpose and normalize
// [h, w, c] -> [c, h, w]
// 0...255 -> 0...1
// BGR -> RGB
__global__ void blobFromImage(uint8_t *input, float *output, int h, int w, int c, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < h && y < w && z < c)
    {
        for (int ibatch = 0; ibatch < n; ibatch++)
        {
            int input_idx = x * (c * w) + y * c + (2 - z);
            int output_idx = ibatch * (w * h * c) + z * (w * h) + x * w + y;
            output[output_idx] = input[input_idx] / 255.f;
        }
    }
}

// preprocess for detection
void Detect::preprocess(std::vector<FrameInfo> &images)
{
    int batch_size = images.size();
    // LOG_ASSERT(batch_size) << " images is empty";
    if (batch_size == 0)
    {
        LOG(WARNING) << " images is empty";
        return;
    }

    uint8_t *d_ptr_src;                                                 // device pointer for src image
    uint8_t *d_ptr_dst;                                                 // device pointer for dst image
    int src_w;                                                          // src image width
    int src_h;                                                          // src image height
    int dst_w = this->input_width;                                      // dst image width
    int dst_h = this->input_height;                                     // dst image height
    size_t src_size;                                                    // src image size
    size_t dst_size = batch_size * dst_w * dst_h * 3 * sizeof(uint8_t); // dst image size

    CUDA_CHECK(hipMalloc((void**)&d_ptr_dst, dst_size));

    for (int ibatch = 0; ibatch < batch_size; ibatch++)
    {
        FrameInfo src = images[ibatch];
        src_w = src.frame.cols;
        src_h = src.frame.rows;
        src_size = src_w * src_h * 3 * sizeof(uint8_t);

        // LOG(INFO) << "batch: " << ibatch << ", src_w: " << src_w << ", src_h: " << src_h << ", dst_w: " << dst_w << ", dst_h: " << dst_h;

        CUDA_CHECK(hipMalloc((void**)&d_ptr_src, src_size));
        CUDA_CHECK(hipMemcpy(d_ptr_src, src.frame.data, src_size, hipMemcpyHostToDevice));

        // compute affine tranformation matrix
        (this->affine_matrix).compute(cv::Size(src_w, src_h), cv::Size(dst_w, dst_h));

        dim3 block1(32, 32);
        dim3 grid1((dst_w + block1.x - 1) / block1.x, (dst_h + block1.y - 1) / block1.y);

        // LOG(INFO) << "warp_affine kernel launched with "
        //           << grid1.x << "x" << grid1.y << "x" << grid1.z << " blocks of "
        //           << block1.x << "x" << block1.y << "x" << block1.z << " threads, "
        //           << "src_w: " << src_w << ", src_h: " << src_h
        //           << ", dst_w: " << dst_w << ", dst_h: " << dst_h;

        // do letterbox transformation on src image
        // src: [src_h, src_w, 3], dst: [dst_h, dst_w, 3]
        warp_affine<<<grid1, block1>>>(
            d_ptr_src, src_w, src_h,
            d_ptr_dst, dst_w, dst_h,
            114, this->affine_matrix, ibatch);

        CUDA_CHECK(hipFree(d_ptr_src));
    }

    // // warp affine test code, currently no bug
    // if (images[0].info == "water crops")
    // {
    //     cv::imwrite("src.jpg", images[0].frame);
    //     view_device_input_img_batch(d_ptr_dst, batch_size, 3, dst_h, dst_w, "dst");
    //     // LOG_ASSERT(0) << "stop here";
    // }

    dim3 block2(16, 16, 4);
    dim3 grid2((dst_w + block2.x - 1) / block2.x, (dst_h + block2.y - 1) / block2.y, (3 + block2.z - 1) / block2.z);

    // LOG(INFO) << "blobFromImage kernel launched with "
    //           << grid2.x << "x" << grid2.y << "x" << grid2.z << " blocks of "
    //           << block2.x << "x" << block2.y << "x" << block2.z << " threads";

    blobFromImage<<<grid2, block2>>>(
        d_ptr_dst, (float *)this->device_ptrs[0],
        dst_h, dst_w, 3, batch_size);

    CUDA_CHECK(hipFree(d_ptr_dst));

    // blobFromImage test code, currently no bug
    // view_device_batch_img((float*)this->device_ptrs[0], batch_size, 3, this->input_width, this->input_height, "input");
    // LOG_ASSERT(0) << "stop here";
}

// preprocess for segmentation
void Segment::preprocess(std::vector<CropInfo> &crops)
{
    int batch_size = crops.size();
    LOG_ASSERT(batch_size) << "crops is empty";

    uint8_t *d_ptr;
    int w = this->input_width;
    int h = this->input_height;

    size_t size = w * h * 3 * sizeof(uint8_t);

    CUDA_CHECK(hipMalloc((uint8_t **)&d_ptr, batch_size * size));

    int ibatch = 0;
    for (auto crop_info : crops)
    {
        cv::resize(crop_info.crop, crop_info.crop, cv::Size(this->input_width, this->input_height));
        // LOG(INFO) << "crop size" << crop_info.crop.size();
        CUDA_CHECK(hipMemcpy(d_ptr + ibatch * w * h * 3, crop_info.crop.data, size, hipMemcpyHostToDevice));
        ibatch++;
    }

    dim3 block(16, 16, 3);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (3 + block.z - 1) / block.z);

    // LOG(INFO) << "blobFromImage kernel launched with "
    //           << grid.x << "x" << grid.y << "x" << grid.z << " blocks of "
    //           << block.x << "x" << block.y << "x" << block.z << " threads";

    blobFromImage<<<grid, block>>>(
        d_ptr, (float *)this->device_ptrs[0],
        h, w, 3, batch_size);

    // blobFromImage test code, currently no bug
    // view_device_batch_img((float*)this->device_ptrs[0], batch_size, 3, this->input_width, this->input_height, "input_seg");
    // LOG_ASSERT(0) << "stop here";

    CUDA_CHECK(hipFree(d_ptr));
}